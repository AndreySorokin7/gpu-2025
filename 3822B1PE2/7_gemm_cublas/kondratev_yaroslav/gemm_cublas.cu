#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    if (n == 0) return {};

    size_t matrixSize = n * n;
    std::vector<float> c(matrixSize, 0.0f);
    size_t bytes = matrixSize * sizeof(float);    
    float *d_a{}, *d_b{}, *d_c{};
    constexpr float alpha = 1.0f;
    constexpr float beta = 0.0f;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n, &alpha, d_b, n,
                d_a, n, &beta, d_c, n);
    hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    hipblasDestroy(handle);
    return c;
}