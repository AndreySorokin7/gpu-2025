#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <stdexcept>

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        throw std::runtime_error("CUDA error");
    }
}

void checkCublasError(hipblasStatus_t status, const char* msg) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS error: " << msg << " - " << status << std::endl;
        throw std::runtime_error("cuBLAS error");
    }
}

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
    const std::vector<float>& b,
    int n) {

    if (a.size() != static_cast<size_t>(n * n) ||
        b.size() != static_cast<size_t>(n * n)) {
        throw std::invalid_argument("Matrix sizes do not match n*n");
    }

    size_t bytes = n * n * sizeof(float);
    std::vector<float> c(n * n);

    if (n == 0) return c;

    float* d_a = nullptr, * d_b = nullptr, * d_c = nullptr;

    checkCudaError(hipMalloc(&d_a, bytes), "hipMalloc d_a");
    checkCudaError(hipMalloc(&d_b, bytes), "hipMalloc d_b");
    checkCudaError(hipMalloc(&d_c, bytes), "hipMalloc d_c");

    hipblasHandle_t handle;
    checkCublasError(hipblasCreate(&handle), "hipblasCreate");

    checkCudaError(hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice), "Copy a to device");
    checkCudaError(hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice), "Copy b to device");
    const float alpha = 1.0f;
    const float beta = 0.0f;

    checkCublasError(
        hipblasSgemm(handle,
            HIPBLAS_OP_N,
            HIPBLAS_OP_N,
            n,
            n,
            n,
            &alpha,
            d_b, n,
            d_a, n,
            &beta,
            d_c, n),
        "hipblasSgemm"
    );

    checkCudaError(hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost), "Copy c from device");

    hipblasDestroy(handle);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}