#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdexcept>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    std::vector<float> c(n * n);
    
    float *d_A, *d_B, *d_C;
    size_t size = n * n * sizeof(float);
    
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    hipblasSetMatrix(n, n, sizeof(float), a.data(), n, d_A, n);
    hipblasSetMatrix(n, n, sizeof(float), b.data(), n, d_B, n);
    
    constexpr float alpha = 1.0f;
    constexpr float beta = 0.0f;

    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_B, n,
                d_A, n,
                &beta,
                d_C, n);
    
    hipblasGetMatrix(n, n, sizeof(float), d_C, n, c.data(), n);
    
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return c;
}
