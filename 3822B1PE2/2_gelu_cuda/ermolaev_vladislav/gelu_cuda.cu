#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include "gelu_cuda.h"

__constant__ float sqrt_2_pi;
__constant__ float coeff = 0.044715f;

__global__ void gelu_kernel(const float* input, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = input[idx];
        float x3 = x * x * x;
        float arg = sqrt_2_pi * (x + coeff * x3);
        float exp2z = expf(2.0f * arg);
        float tanh_val = (exp2z - 1.0f) / (exp2z + 1.0f);
        output[idx] = 0.5f * x * (1.0f + tanh_val);
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    int n = input.size();
    std::vector<float> output(n);

    float val1 = std::sqrt(2.0f / acosf(-1.0f));
    hipMemcpyToSymbol(HIP_SYMBOL(sqrt_2_pi), &val1, sizeof(float));

    float *d_input, *d_output;
    hipMalloc(&d_input, n * sizeof(float));
    hipMalloc(&d_output, n * sizeof(float));

    hipMemcpy(d_input, input.data(), n * sizeof(float), hipMemcpyHostToDevice);

    int minGridSize, blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, gelu_kernel, 0, 0);

    int numBlocks = (n + blockSize - 1) / blockSize;
    gelu_kernel<<<numBlocks, blockSize>>>(d_input, d_output, n);
    hipDeviceSynchronize();

    hipMemcpy(output.data(), d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}