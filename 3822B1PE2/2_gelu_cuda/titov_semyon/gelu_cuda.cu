#include "hip/hip_runtime.h"
#include "gelu_cuda.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>

__global__ void GeluKernelFast(const float* input, float* output, int n) {
    const float a = 1.702f;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x = input[idx];
        output[idx] = x * (0.5f + 0.5f * tanhf(x * a * 0.5f));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    int n = input.size();
    std::vector<float> output(n);
    
    float *d_input, *d_output;
    hipMalloc(&d_input, n * sizeof(float));
    hipMalloc(&d_output, n * sizeof(float));
    
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    hipMemcpyAsync(d_input, input.data(), n * sizeof(float), hipMemcpyHostToDevice, stream);
    
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    
    GeluKernelFast<<<numBlocks, blockSize, 0, stream>>>(d_input, d_output, n);
    hipMemcpyAsync(output.data(), d_output, n * sizeof(float), hipMemcpyDeviceToHost, stream);
    
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
    
    hipFree(d_input);
    hipFree(d_output);
    
    return output;
}