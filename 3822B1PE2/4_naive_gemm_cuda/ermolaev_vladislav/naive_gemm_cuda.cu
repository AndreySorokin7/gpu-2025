#include "hip/hip_runtime.h"
#include "naive_gemm_cuda.h"
#include <vector>
#include <hip/hip_runtime.h>
#include <cmath>

__global__ void naiveGemmKernel(const float* a, const float* b, float* c, int n) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= n && j >= n) return;

    float sum = 0.0f;
    for (int k = 0; k < n; ++k) {
        sum += a[i * n + k] * b[k * n + j];
    }
    c[i * n + j] = sum;
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) {
    std::vector<float> c(n * n);
    
    float *d_a, *d_b, *d_c;
    size_t size = n * n * sizeof(float);
    
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    
    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int maxThreadsPerBlock = prop.maxThreadsPerBlock;
    
    int sqrtMax = static_cast<int>(sqrtf(maxThreadsPerBlock));
    int blockSize = 1;
    while (blockSize * 2 <= sqrtMax) {
        blockSize <<= 1;
    }
    
    dim3 threadsPerBlock(blockSize, blockSize);
    dim3 numBlocks((n + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (n + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    naiveGemmKernel<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();

    hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return c;
}