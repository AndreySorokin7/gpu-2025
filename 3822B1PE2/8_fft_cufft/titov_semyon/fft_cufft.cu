#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <stdexcept>
#include <cmath>

__global__ void normalize_kernel(hipfftComplex* data, float scale, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx].x *= scale;
        data[idx].y *= scale;
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        throw std::runtime_error("CUDA error");
    }
}

void checkCufftError(hipfftResult_t result, const char* msg) {
    if (result != HIPFFT_SUCCESS) {
        std::cerr << "cuFFT error: " << msg << " - " << result << std::endl;
        throw std::runtime_error("cuFFT error");
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    if (input.empty()) {
        return std::vector<float>();
    }
    if (batch <= 0) {
        throw std::invalid_argument("Batch must be positive");
    }
    if (input.size() % (2 * batch) != 0) {
        throw std::invalid_argument("Input size must be divisible by 2 * batch");
    }
    int n = input.size() / (2 * batch);
    if (n <= 0) {
        throw std::invalid_argument("Invalid signal length");
    }

    size_t complex_size = n * batch;
    size_t bytes = complex_size * sizeof(hipfftComplex);

    std::vector<float> output(input.size());

    hipfftComplex* d_input = nullptr, * d_fft = nullptr, * d_ifft = nullptr;

    checkCudaError(hipMalloc(&d_input, bytes), "hipMalloc d_input");
    checkCudaError(hipMalloc(&d_fft, bytes), "hipMalloc d_fft");
    checkCudaError(hipMalloc(&d_ifft, bytes), "hipMalloc d_ifft");
    checkCudaError(hipMemcpy(d_input, input.data(), bytes, hipMemcpyHostToDevice), "Copy input to device");
    hipfftHandle plan_forward;
    checkCufftError(hipfftPlan1d(&plan_forward, n, HIPFFT_C2C, batch), "hipfftPlan1d forward");

    hipfftHandle plan_inverse;
    checkCufftError(hipfftPlan1d(&plan_inverse, n, HIPFFT_C2C, batch), "hipfftPlan1d inverse");
    checkCufftError(hipfftExecC2C(plan_forward, d_input, d_fft, HIPFFT_FORWARD), "hipfftExecC2C forward");
    checkCufftError(hipfftExecC2C(plan_inverse, d_fft, d_ifft, HIPFFT_BACKWARD), "hipfftExecC2C inverse");

    float scale = 1.0f / n;
    int total_elements = complex_size;

    int blockSize = 256;
    int numBlocks = (total_elements + blockSize - 1) / blockSize;

    normalize_kernel << <numBlocks, blockSize >> > (d_ifft, scale, total_elements);

    checkCudaError(hipGetLastError(), "Normalization kernel");
    checkCudaError(hipDeviceSynchronize(), "Kernel synchronization");

    checkCudaError(hipMemcpy(output.data(), d_ifft, bytes, hipMemcpyDeviceToHost), "Copy result from device");

    hipfftDestroy(plan_forward);
    hipfftDestroy(plan_inverse);
    hipFree(d_input);
    hipFree(d_fft);
    hipFree(d_ifft);

    return output;
}