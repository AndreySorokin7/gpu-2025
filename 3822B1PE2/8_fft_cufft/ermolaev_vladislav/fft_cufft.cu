#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdexcept>

__global__ void normalizeKernel(hipfftComplex* data, int size, float scale) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx].x *= scale;
        data[idx].y *= scale;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int n = input.size() / (2 * batch);
    int total_elements = n * batch;
    
    std::vector<float> output(input.size());
    
    hipfftComplex *d_data;
    size_t size = total_elements * sizeof(hipfftComplex);
    hipMalloc(&d_data, size);

    hipMemcpy(d_data, input.data(), input.size() * sizeof(float), hipMemcpyHostToDevice);
    
    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);
    
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);
    
    int minGridSize, blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, normalizeKernel, 0, 0);
    
    int numBlocks = (total_elements + blockSize - 1) / blockSize;
    float scale = 1.0f / n;
    normalizeKernel<<<numBlocks, blockSize>>>(d_data, total_elements, scale);

    hipMemcpy(output.data(), d_data, input.size() * sizeof(float), hipMemcpyDeviceToHost);
    
    hipfftDestroy(plan);
    hipFree(d_data);
    
    return output;
}