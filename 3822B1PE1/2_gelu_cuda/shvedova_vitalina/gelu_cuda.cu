#include "hip/hip_runtime.h"
#include "gelu_cuda.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
#include <stdexcept>

__device__ __forceinline__ float gelu_single(float x) {
    const float alpha = sqrtf(2.0f / 3.14159265f);
    const float c = 0.044715f;
    float x3 = x * x * x;
    float t = alpha * (x + c * x3);
    float tanh_t = tanhf(t);
    return 0.5f * x * (1.0f + tanh_t);
}

__global__ void gelu_kernel(const float* __restrict__ in,
                            float* __restrict__ out,
                            int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = gelu_single(in[idx]);
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    int n = static_cast<int>(input.size());
    if (n == 0) return {};

    size_t bytes = n * sizeof(float);
    float *d_in = nullptr, *d_out = nullptr;

    if (hipMalloc(&d_in, bytes) != hipSuccess ||
        hipMalloc(&d_out, bytes) != hipSuccess) {
        throw std::runtime_error("CUDA malloc failed");
    }

    hipMemcpy(d_in, input.data(), bytes, hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    gelu_kernel<<<gridSize, blockSize>>>(d_in, d_out, n);

    hipDeviceSynchronize();

    std::vector<float> output(n);
    hipMemcpy(output.data(), d_out, bytes, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

    return output;
}
