#include "hip/hip_runtime.h"
//
// Created by korablev-vm on 27.09.2025.
//

#include "gelu_cuda.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

constexpr float SQRT_2_OVER_PI = 0.7978845608f;
constexpr float GELU_COEFF     = 0.044715f;

__global__ void GeluCUDA_kernel(float* data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float x  = data[idx];
        float x2 = x * x;
        float inner = SQRT_2_OVER_PI * (x + GELU_COEFF * x * x2);

        data[idx] = 0.5f * x * (1.0f + tanhf(inner));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    int threadsPerBlock = deviceProp.maxThreadsPerBlock;
    int blocksNum = (input.size() + threadsPerBlock - 1) / threadsPerBlock;

    std::vector<float> output(input);

    float* d_ptr = nullptr;
    hipMalloc(&d_ptr, sizeof(float) * input.size());
    hipMemcpy(d_ptr, output.data(), sizeof(float) * input.size(), hipMemcpyHostToDevice);

    GeluCUDA_kernel<<<blocksNum, threadsPerBlock>>>(d_ptr, static_cast<int>(input.size()));
    hipDeviceSynchronize();

    hipMemcpy(output.data(), d_ptr, sizeof(float) * input.size(), hipMemcpyDeviceToHost);
    hipFree(d_ptr);

    return output;
}
