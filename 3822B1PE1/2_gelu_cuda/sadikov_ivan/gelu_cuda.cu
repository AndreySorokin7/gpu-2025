#include "hip/hip_runtime.h"
#include "gelu_cuda.h"

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void Calculate(const float* inputData, float* result, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size)
    {
        result[index] = 0.5f * inputData[index] * (1.0f  + 
            tanhf(0.7978f * (inputData[index] + 0.044715f * inputData[index] * inputData[index] * inputData[index])));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) 
{
    auto size {input.size() * sizeof(float)};
    constexpr int threadsCount {256};
    int blocksCount {(static_cast<int>(input.size()) + threadsCount - 1) / threadsCount};

    std::vector<float> result(input.size());

    float * inputData;
    float * resultFromCUDA;

    hipMalloc(&inputData, size);
    hipMalloc(&resultFromCUDA, size);

    hipMemcpy(inputData, input.data(), size, hipMemcpyHostToDevice);

    Calculate<<<blocksCount, threadsCount>>>(inputData, resultFromCUDA, static_cast<int>(input.size()));

    hipMemcpy(resultFromCUDA, result.data(), size, hipMemcpyDeviceToHost);
    
    hipFree(inputData);
    hipFree(resultFromCUDA);
    
    return result;
}