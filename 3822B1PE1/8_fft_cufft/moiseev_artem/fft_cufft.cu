#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

__global__ void normalize_kernel(hipfftComplex* data, int n, int total_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_elements) {
        float scale = 1.0f / static_cast<float>(n);
        data[idx].x *= scale;
        data[idx].y *= scale;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    size_t total_size = input.size();
    if (total_size % (2 * batch) != 0) {
        throw std::invalid_argument("Invalid Input size");
    }
    int n = static_cast<int>(total_size / (2 * batch));

    hipfftComplex* d_data = nullptr;
    size_t complex_size = n * batch * sizeof(hipfftComplex);

    hipMalloc(&d_data, complex_size);
    hipMemcpy(d_data, input.data(), complex_size, hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);

    int total_elements = n * batch;
    int block_size = 256;
    int grid_size = (total_elements + block_size - 1) / block_size;
    normalize_kernel<<<grid_size, block_size>>>(d_data, n, total_elements);

    std::vector<float> result(total_size);
    hipMemcpy(result.data(), d_data, complex_size, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_data);

    return result;
}