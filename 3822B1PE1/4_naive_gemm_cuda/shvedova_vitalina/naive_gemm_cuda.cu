#include "hip/hip_runtime.h"
#include "naive_gemm_cuda.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <vector>
#include <iostream>

__global__ void gemm_naive_kernel(const float* a,
                                  const float* b,
                                  float* c,
                                  int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) {
    if (a.size() != static_cast<size_t>(n * n) ||
        b.size() != static_cast<size_t>(n * n)) {
        throw std::invalid_argument("Matrix sizes do not match n*n");
    }

    size_t bytes = n * n * sizeof(float);
    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;

    if (hipMalloc(&d_a, bytes) != hipSuccess ||
        hipMalloc(&d_b, bytes) != hipSuccess ||
        hipMalloc(&d_c, bytes) != hipSuccess) {
        throw std::runtime_error("CUDA malloc failed");
    }

    if (hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice) != hipSuccess ||
        hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice) != hipSuccess) {
        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        throw std::runtime_error("CUDA memcpy (H2D) failed");
    }

    dim3 blockSize(16, 16);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x,
                  (n + blockSize.y - 1) / blockSize.y);

    gemm_naive_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    if (hipGetLastError() != hipSuccess ||
        hipDeviceSynchronize() != hipSuccess) {
        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        throw std::runtime_error("CUDA kernel execution failed");
    }

    std::vector<float> c(n * n);
    if (hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost) != hipSuccess) {
        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        throw std::runtime_error("CUDA memcpy (D2H) failed");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}

