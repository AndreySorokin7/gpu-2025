#include "hip/hip_runtime.h"
#include "naive_gemm_cuda.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel(const float* __restrict__ a, const float* __restrict__ b, float* __restrict__ c, size_t n) {
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  size_t y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= n || y >= n) {
    return;
  }

  b = &b[x];
  const float4* a4 = reinterpret_cast<const float4*>(&a[y*n]);

  for (size_t k = 0; k < (n >> 2); ++k) {
    float4 a = a4[k];

    c[y * n + x] += a.x * b[k * 4 * n] 
                  + a.y * b[(k * 4 + 1) * n] 
                  + a.z * b[(k * 4 + 2) * n] 
                  + a.w * b[(k * 4 + 3) * n];
  }
}

std::vector<float> NaiveGemmCUDA(
  const std::vector<float>& a,
  const std::vector<float>& b,
  int n
) {
  size_t size = n * n * sizeof(float);
  float *device_a, *device_b, *device_c;
  std::vector<float> c(n * n, 0);

  hipMalloc(&device_a, size);
  hipMalloc(&device_b, size);
  hipMalloc(&device_c, size);

  hipMemcpy(device_a, a.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(device_b, b.data(), size, hipMemcpyHostToDevice);

  constexpr int block_size = 32;
  dim3 block_dim(block_size, block_size);
  dim3 num_blocks((n + block_size - 1) / block_size, (n + block_size - 1) / block_size);

  kernel<<<num_blocks, block_dim>>>(device_a, device_b, device_c, n);

  hipMemcpy(c.data(), device_c, size, hipMemcpyDeviceToHost);

  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);

  return c;
}
