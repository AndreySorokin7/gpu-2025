#include "hip/hip_runtime.h"
#include "naive_gemm_cuda.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <iostream>

__global__ void gemm_naive_kernel(const float* a, const float* b, float* c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        throw std::runtime_error("CUDA error");
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) {
                                    
    if (a.size() != static_cast<size_t>(n * n) ||
        b.size() != static_cast<size_t>(n * n)) {
        throw std::invalid_argument("Matrix sizes do not match n*n");
    }

    size_t bytes = n * n * sizeof(float);
    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;

    checkCudaError(hipMalloc(&d_a, bytes), "hipMalloc d_a");
    checkCudaError(hipMalloc(&d_b, bytes), "hipMalloc d_b");
    checkCudaError(hipMalloc(&d_c, bytes), "hipMalloc d_c");

    checkCudaError(hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice), "Copy a to device");
    checkCudaError(hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice), "Copy b to device");

    dim3 blockSize(16, 16);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x,
                  (n + blockSize.y - 1) / blockSize.y);

    gemm_naive_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    checkCudaError(hipGetLastError(), "Kernel launch");
    checkCudaError(hipDeviceSynchronize(), "Kernel execution");

    std::vector<float> c(n * n);
    checkCudaError(hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost), "Copy c from device");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}