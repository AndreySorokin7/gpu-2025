#include "gemm_cublas.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

struct Data
{
  const float alphaMultiplier {1.0f};
  const float bettaMultiplier {0.0f};
  int size {0};
};

void multiplicator(const float* first, 
                   const float* second,
                   float* result,
                   Data& data)
{
   hipblasHandle_t handle;
   hipblasCreate(&handle);

   int size = data.size;
   hipblasSgemm(handle, 
               HIPBLAS_OP_T, 
               HIPBLAS_OP_T, 
               size,
               size, 
               size,
               &data.alphaMultiplier, 
               first, 
               size, 
               second, 
               size, 
               &data.bettaMultiplier, 
               result, 
               size);
   hipblasDestroy(handle);
}

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) 
{
  Data data;
  data.size = n;

  std::vector<float> result(n*n);
  int bytesCount = n * n * static_cast<int>(sizeof(float));

  float* firstMatrix;
  float* secondMatrix;
  float* resultMatrix;

  hipMalloc(&firstMatrix, bytesCount);
  hipMalloc(&secondMatrix, bytesCount);
  hipMalloc(&resultMatrix, bytesCount);

  hipMemcpy(firstMatrix, a.data(), bytesCount, hipMemcpyHostToDevice);
  hipMemcpy(secondMatrix, b.data(), bytesCount, hipMemcpyHostToDevice);

  multiplicator(firstMatrix, secondMatrix, resultMatrix, data);

  hipMemcpy(result.data(), resultMatrix, bytesCount, hipMemcpyDeviceToHost);

  hipFree(firstMatrix);
  hipFree(secondMatrix);
  hipFree(resultMatrix);

  return result;
}
