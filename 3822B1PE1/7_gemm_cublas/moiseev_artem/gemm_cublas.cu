#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdexcept>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
    const size_t size = n * n * sizeof(float);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_b, n,
                d_a, n,
                &beta,
                d_c, n);

    std::vector<float> result(n * n);
    hipMemcpy(result.data(), d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(handle);

    return result;
}