#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    if (n <= 0) {
        throw std::invalid_argument("Matrix size n must be positive");
    }
    const size_t matrix_size = n * n;
    if (a.size() != matrix_size || b.size() != matrix_size) {
        throw std::invalid_argument("Matrix sizes do not match n*n");
    }

    const size_t bytes = matrix_size * sizeof(float);
    

    std::vector<float> c(matrix_size);

    float* d_a = nullptr;
    float* d_b = nullptr; 
    float* d_c = nullptr;
    
    hipError_t cuda_status;
    cuda_status = hipMalloc(&d_a, bytes);
    if (cuda_status != hipSuccess) throw std::runtime_error("hipMalloc failed for d_a");
    
    cuda_status = hipMalloc(&d_b, bytes);
    if (cuda_status != hipSuccess) {
        hipFree(d_a);
        throw std::runtime_error("hipMalloc failed for d_b");
    }
    
    cuda_status = hipMalloc(&d_c, bytes);
    if (cuda_status != hipSuccess) {
        hipFree(d_a);
        hipFree(d_b);
        throw std::runtime_error("hipMalloc failed for d_c");
    }

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipMemcpyAsync(d_a, a.data(), bytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, b.data(), bytes, hipMemcpyHostToDevice, stream);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, stream);

    hipStreamSynchronize(stream);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle,
                HIPBLAS_OP_N, 
                HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_a, n,
                d_b, n,  
                &beta,
                d_c, n);

    hipMemcpyAsync(c.data(), d_c, bytes, hipMemcpyDeviceToHost, stream);

    hipStreamSynchronize(stream);

    hipblasDestroy(handle);
    hipStreamDestroy(stream);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}