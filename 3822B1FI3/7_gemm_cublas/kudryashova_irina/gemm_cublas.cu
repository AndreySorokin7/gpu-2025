#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {


    const size_t bytes = static_cast<size_t>(n) * n * sizeof(float);
    std::vector<float> host_c(n * n);

    hipStream_t compute_stream;
    hipStreamCreateWithFlags(&compute_stream, hipStreamNonBlocking);

    hipHostRegister(const_cast<float*>(a.data()), bytes, 0);
    hipHostRegister(const_cast<float*>(b.data()), bytes, 0);
    hipHostRegister(host_c.data(), bytes, 0);

    float *dev_a = nullptr, *dev_b = nullptr, *dev_c = nullptr;
    hipMalloc(&dev_a, bytes);
    hipMalloc(&dev_b, bytes);
    hipMalloc(&dev_c, bytes);

    hipMemcpyAsync(dev_a, a.data(), bytes, hipMemcpyHostToDevice, compute_stream);
    hipMemcpyAsync(dev_b, b.data(), bytes, hipMemcpyHostToDevice, compute_stream);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, compute_stream);
    (void)hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);

    const float one  = 1.0f;
    const float zero = 0.0f;
    int side = n;

    hipblasSgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        n,  // m
        n,  // n
        n,  // k
        &one,
        dev_b, side, 
        dev_a, side,
        &zero,
        dev_c, side
    );

    hipMemcpyAsync(host_c.data(), dev_c, bytes, hipMemcpyDeviceToHost, compute_stream);
    hipStreamSynchronize(compute_stream);

    hipblasDestroy(handle);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    hipHostUnregister(const_cast<float*>(a.data()));
    hipHostUnregister(const_cast<float*>(b.data()));
    hipHostUnregister(host_c.data());
    hipStreamDestroy(compute_stream);

    return host_c;
}