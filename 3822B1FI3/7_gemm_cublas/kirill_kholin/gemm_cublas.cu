#include "gemm_cublas.h"
#include <hipblas.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    std::vector<float> c(n * n);

    float *d_a;
    float *d_b;
    float *d_c;

    size_t bytes = n * n * sizeof(float);

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipMemcpyAsync(d_a, a.data(), bytes, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_b, b.data(), bytes, hipMemcpyHostToDevice, stream);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSetStream(handle, stream);

    hipStreamSynchronize(stream);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_b, n,
                d_a, n,
                &beta,
                d_c, n);

    hipMemcpyAsync(c.data(), d_c, bytes, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    hipblasDestroy(handle);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipStreamDestroy(stream);

    return c;
}