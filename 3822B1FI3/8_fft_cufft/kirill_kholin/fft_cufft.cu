#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>

__global__ void normalize_kernel(hipfftComplex* data, int n, int total_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_elements) {
        data[idx].x /= n;
        data[idx].y /= n;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int n = input.size() / (2 * batch);
    size_t complex_size = n * batch * sizeof(hipfftComplex);
    
    std::vector<float> output(input.size());
    hipfftComplex* d_data;
    
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    hipMalloc(&d_data, complex_size);
    hipMemcpyAsync(d_data, input.data(), complex_size, hipMemcpyHostToDevice, stream);
    
    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);
    hipfftSetStream(plan, stream);
    
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);
    
    int threads = 256;
    int blocks = (n * batch + threads - 1) / threads;
    normalize_kernel<<<blocks, threads, 0, stream>>>(d_data, n, n * batch);
    
    hipMemcpyAsync(output.data(), d_data, complex_size, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);
    
    hipfftDestroy(plan);
    hipFree(d_data);
    hipStreamDestroy(stream);
    
    return output;
}