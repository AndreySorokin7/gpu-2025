#include "gemm_cublas.h"

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    size_t N = static_cast<size_t>(n);

    std::vector C(N * N, 0.0f);

    const size_t bytes = N * N * sizeof(float);

    float* D_a = nullptr, * D_b = nullptr, * D_c = nullptr;

    const float alpha = 1.0f, beta = 0.0f;

    hipMalloc(&D_a, bytes);
    hipMalloc(&D_b, bytes);
    hipMalloc(&D_c, bytes);

    hipMemcpy(D_a, a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(D_b, b.data(), bytes, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSgemm(handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        n, n, n,
        &alpha,
        D_b, n,
        D_a, n,
        &beta,
        D_c, n);

    hipMemcpy(C.data(), D_c, bytes, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);

    hipFree(D_a);
    hipFree(D_b);
    hipFree(D_c);

    return C;
}