#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
    const std::vector<float>& b,
    int n) {
    if (a.size() != static_cast<size_t>(n * n) ||
        b.size() != static_cast<size_t>(n * n)) {
        throw std::invalid_argument("Matrix sizes do not match n*n");
    }

    const size_t bytes = n * n * sizeof(float);
    float* d_a = nullptr, * d_b = nullptr, * d_c = nullptr;

    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_N,
        n, n, n,
        &alpha,
        d_b, n,
        d_a, n,
        &beta,
        d_c, n);

    std::vector<float> c(n * n);
    hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    hipblasDestroy(handle);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}