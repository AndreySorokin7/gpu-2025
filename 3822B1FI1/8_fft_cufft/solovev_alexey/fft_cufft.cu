#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <stdexcept>

__global__ void normalize(hipfftComplex* data, int n, int batch) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = n * batch;
    if (idx < total) {
        data[idx].x /= n;
        data[idx].y /= n;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    if (input.size() % (2 * batch) != 0) {
        throw std::invalid_argument("Input size must be 2 * n * batch");
    }

    int n = input.size() / (2 * batch);
    size_t complexSize = sizeof(hipfftComplex);
    size_t totalSize = n * batch * complexSize;

    std::vector<float> output(input.size());
    hipfftComplex* d_data = nullptr;

    hipMalloc(&d_data, totalSize);
    hipMemcpy(d_data, input.data(), totalSize, hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);

    int threads = 256;
    int blocks = (n * batch + threads - 1) / threads;
    normalize << <blocks, threads >> > (d_data, n, batch);

    hipMemcpy(output.data(), d_data, totalSize, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(d_data);

    return output;
}