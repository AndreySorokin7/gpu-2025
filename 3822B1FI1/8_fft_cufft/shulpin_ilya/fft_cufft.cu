#include "hip/hip_runtime.h"
#include "fft_cufft.h"

__global__ void normalize(hipfftComplex* data, int total, float scale) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total) {
        data[idx].x *= scale;
        data[idx].y *= scale;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    size_t total = input.size();
    size_t N = total / (2 * static_cast<size_t>(batch));

    size_t total_complex = N * static_cast<size_t>(batch);
    int total_batch = static_cast<int>(total_complex);

    size_t bytes = total_complex * sizeof(hipfftComplex);
    std::vector<float> out(total);

    float scale = 1.0f / static_cast<float>(N);
    int block = 256;
    int grid = (total_batch + block - 1) / block;

    hipfftComplex* data = nullptr;
    hipMalloc(&data, bytes);
    hipMemcpy(data, input.data(), bytes, hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, static_cast<int>(N), HIPFFT_C2C, batch);

    hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
    hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD);

    normalize<<<grid, block>>>(data, total_batch, scale);
    hipDeviceSynchronize();

    hipMemcpy(out.data(), data, bytes, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(data);

    return out;
}