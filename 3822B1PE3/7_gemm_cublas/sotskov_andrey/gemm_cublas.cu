#include "gemm_cublas.h"
#include <hipblas.h>

std::vector<float> GemmCUBLAS(const std::vector<float> &matrixA,
                              const std::vector<float> &matrixB, int matrixSize)
{
    std::vector<float> resultMatrix(matrixSize * matrixSize);

    float *deviceMatrixA = nullptr;
    float *deviceMatrixB = nullptr;
    float *deviceMatrixC = nullptr;

    const size_t matrixBytes = matrixSize * matrixSize * sizeof(float);

    hipMalloc(&deviceMatrixA, matrixBytes);
    hipMalloc(&deviceMatrixB, matrixBytes);
    hipMalloc(&deviceMatrixC, matrixBytes);

    hipMemcpy(deviceMatrixA, matrixA.data(), matrixBytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceMatrixB, matrixB.data(), matrixBytes, hipMemcpyHostToDevice);

    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(cublasHandle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                matrixSize, matrixSize, matrixSize,
                &alpha,
                deviceMatrixB, matrixSize,
                deviceMatrixA, matrixSize,
                &beta,
                deviceMatrixC, matrixSize);

    hipMemcpy(resultMatrix.data(), deviceMatrixC, matrixBytes, hipMemcpyDeviceToHost);

    hipblasDestroy(cublasHandle);
    hipFree(deviceMatrixA);
    hipFree(deviceMatrixB);
    hipFree(deviceMatrixC);

    return resultMatrix;
}