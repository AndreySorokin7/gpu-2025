#include "hip/hip_runtime.h"
#include "block_gemm_cuda.h"

#include <hip/hip_runtime.h>
#include <vector>
#include <stdexcept>
#include <cstring>
#include <iostream>

#define CHECK_CUDA(call) do {                              \
    hipError_t err = (call);                              \
    if (err != hipSuccess) {                              \
        throw std::runtime_error(hipGetErrorString(err)); \
    }                                                      \
} while(0)

constexpr int TILE = 16;

__global__ void block_gemm_kernel(const float* __restrict__ A,
                                  const float* __restrict__ B,
                                  float* __restrict__ C,
                                  int n) {
    __shared__ float sA[TILE][TILE];
    __shared__ float sB[TILE][TILE];

    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    int row = blockRow * TILE + threadIdx.y;
    int col = blockCol * TILE + threadIdx.x;

    float acc = 0.0f;

    for (int m = 0; m < n; m += TILE) {
        int a_col = m + threadIdx.x;
        if (row < n && a_col < n) {
            sA[threadIdx.y][threadIdx.x] = A[row * n + a_col];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        int b_row = m + threadIdx.y;
        if (b_row < n && col < n) {
            sB[threadIdx.y][threadIdx.x] = B[b_row * n + col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE; ++k) {
            acc += sA[threadIdx.y][k] * sB[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < n && col < n) {
        C[row * n + col] = acc;
    }
}

std::vector<float> BlockGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) {
    if (n <= 0) return {};
    if (a.size() != static_cast<size_t>(n) * n || b.size() != static_cast<size_t>(n) * n) {
        throw std::invalid_argument("Input sizes must be n*n");
    }

    size_t elems = static_cast<size_t>(n) * n;
    size_t bytes = elems * sizeof(float);
    std::vector<float> out(elems);

    hipStream_t stream;
    CHECK_CUDA(hipStreamCreate(&stream));

    float *dA = nullptr, *dB = nullptr, *dC = nullptr;
    CHECK_CUDA(hipMalloc(&dA, bytes));
    CHECK_CUDA(hipMalloc(&dB, bytes));
    CHECK_CUDA(hipMalloc(&dC, bytes));

    float *hA = nullptr, *hB = nullptr, *hC = nullptr;
    CHECK_CUDA(hipHostMalloc(&hA, bytes));
    CHECK_CUDA(hipHostMalloc(&hB, bytes));
    CHECK_CUDA(hipHostMalloc(&hC, bytes));

    std::memcpy(hA, a.data(), bytes);
    std::memcpy(hB, b.data(), bytes);

    CHECK_CUDA(hipMemcpyAsync(dA, hA, bytes, hipMemcpyHostToDevice, stream));
    CHECK_CUDA(hipMemcpyAsync(dB, hB, bytes, hipMemcpyHostToDevice, stream));

    dim3 blockDim(TILE, TILE);
    dim3 gridDim((n + TILE - 1) / TILE, (n + TILE - 1) / TILE);

    block_gemm_kernel<<<gridDim, blockDim, 0, stream>>>(dA, dB, dC, n);
    CHECK_CUDA(hipGetLastError());

    CHECK_CUDA(hipMemcpyAsync(hC, dC, bytes, hipMemcpyDeviceToHost, stream));

    CHECK_CUDA(hipStreamSynchronize(stream));

    std::memcpy(out.data(), hC, bytes);

    CHECK_CUDA(hipHostFree(hA));
    CHECK_CUDA(hipHostFree(hB));
    CHECK_CUDA(hipHostFree(hC));
    CHECK_CUDA(hipFree(dA));
    CHECK_CUDA(hipFree(dB));
    CHECK_CUDA(hipFree(dC));
    CHECK_CUDA(hipStreamDestroy(stream));

    return out;
}
