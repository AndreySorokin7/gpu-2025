#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "gelu_cuda.h"

__global__ void GeluKernel(const float* input, float* output, size_t size)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= size)
        return;

    constexpr float coefficient1 = 1.595769122f;
    constexpr float coefficient2 = 0.071354816f;

    float value = input[index];
    output[index] = value * (1.0f - 1.0f / (1.0f + __expf(value * (coefficient1 + value * value * coefficient2))));
}

std::vector<float> GeluCUDA(const std::vector<float>& input)
{
    if (input.empty())
        return {};

    size_t data_size = input.size();
    std::vector<float> output(data_size);

    size_t bytes_size = data_size * sizeof(float);
    size_t threads_per_block = 256;
    size_t blocks_count = (data_size + threads_per_block - 1) / threads_per_block;

    float* device_input = nullptr;
    hipMalloc(&device_input, bytes_size);

    float* device_output = nullptr;
    hipMalloc(&device_output, bytes_size);

    hipMemcpy(device_input, input.data(), bytes_size, hipMemcpyHostToDevice);

    GeluKernel<<<blocks_count, threads_per_block>>>(device_input, device_output, data_size);

    hipDeviceSynchronize();
    hipMemcpy(output.data(), device_output, bytes_size, hipMemcpyDeviceToHost);

    hipFree(device_output);
    hipFree(device_input);
    return output;
}