#include "hip/hip_runtime.h"
#include "gelu_cuda.h"

__global__ void GeluKernel(const float* input, float* output, int size, float sqrt_2_pi_m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < size) {
        float x = input[i];
        float x_cubed = x * x * x;
        float expo = expf(sqrt_2_pi_m * (x + 0.044715f * x_cubed));
        output[i] = x * expo / (expo + 1.0f);
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    int size = input.size();
    std::vector<float> output(size);
    
    if (size == 0)
        return output;
    
    const float sqrt_2_pi_m = std::sqrt(2.0f / HIP_PI_F) * 2;
    
    float *dev_input, *dev_output;
    hipMalloc(&dev_input, size * sizeof(float));
    hipMalloc(&dev_output, size * sizeof(float));
    
    hipMemcpy(dev_input, input.data(), size * sizeof(float), hipMemcpyHostToDevice);
    
    int block_size = 256;
    int block_count = (size + block_size - 1) / block_size;
    
    GeluKernel<<<block_count, block_size>>>(dev_input, dev_output, size, sqrt_2_pi_m);
    
    hipMemcpy(output.data(), dev_output, size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(dev_input);
    hipFree(dev_output);
    
    return output;
}