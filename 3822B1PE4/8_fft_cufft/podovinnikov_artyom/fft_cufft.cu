#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "fft_cufft.h"


__global__ void normalize_kernel(hipfftComplex* data, int total, float inv_n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < total) {
    data[i].x *= inv_n;
    data[i].y *= inv_n;
  }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {

  const size_t total_floats = input.size();
  const int n = static_cast<int>(total_floats / (2ULL * batch));
  const int total_complex = n * batch;  
  const size_t bytes = sizeof(hipfftComplex) * total_complex;


  hipfftComplex* d_data = nullptr;
  hipMalloc(&d_data, bytes);

 
  hipMemcpy(d_data, input.data(), bytes, hipMemcpyHostToDevice);


  hipfftHandle plan;
  hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);


  hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
  hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);


  const float inv_n = 1.0f / static_cast<float>(n);
  int block = 256;
  int grid = (total_complex + block - 1) / block;
  normalize_kernel<<<grid, block>>>(d_data, total_complex, inv_n);
  hipDeviceSynchronize();


  std::vector<float> out(total_floats);
  hipMemcpy(out.data(), d_data, bytes, hipMemcpyDeviceToHost);


  hipfftDestroy(plan);
  hipFree(d_data);

  return out;
}
