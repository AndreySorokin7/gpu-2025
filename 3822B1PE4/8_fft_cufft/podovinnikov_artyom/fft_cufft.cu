#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "fft_cufft.h"

// Нормализация на устройстве: делим каждую комплексную точку на n
__global__ void normalize_kernel(hipfftComplex* data, int total, float inv_n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < total) {
    data[i].x *= inv_n;
    data[i].y *= inv_n;
  }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
  // input.size() = 2 * n * batch  → восстановим n
  const size_t total_floats = input.size();
  const int n = static_cast<int>(total_floats / (2ULL * batch));
  const int total_complex = n * batch;  // сколько комплексных чисел всего
  const size_t bytes = sizeof(hipfftComplex) * total_complex;

  // Выделение памяти на девайсе (in-place буфер)
  hipfftComplex* d_data = nullptr;
  hipMalloc(&d_data, bytes);

  // Хост → девайс (в том же интерливинге float2)
  hipMemcpy(d_data, input.data(), bytes, hipMemcpyHostToDevice);

  // План: батч из batch сигналов длины n (C2C, один буфер)
  hipfftHandle plan;
  hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

  // Прямое и обратное преобразования (in-place)
  hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
  hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);

  // Нормализация на устройстве: делим на n
  const float inv_n = 1.0f / static_cast<float>(n);
  int block = 256;
  int grid = (total_complex + block - 1) / block;
  normalize_kernel<<<grid, block>>>(d_data, total_complex, inv_n);
  hipDeviceSynchronize();

  // Девайс → хост
  std::vector<float> out(total_floats);
  hipMemcpy(out.data(), d_data, bytes, hipMemcpyDeviceToHost);

  // Очистка
  hipfftDestroy(plan);
  hipFree(d_data);

  return out;
}
