#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>
#include <stdexcept>

__global__ void normalize_kernel(hipfftComplex* data, int total, float inv_n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < total) {
        data[i].x *= inv_n;
        data[i].y *= inv_n;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    // Проверка входных параметров
    if (batch <= 0) {
        throw std::invalid_argument("Batch must be positive");
    }
    if (input.empty()) {
        throw std::invalid_argument("Input cannot be empty");
    }
    if (input.size() % (2 * batch) != 0) {
        throw std::invalid_argument("Input size must be divisible by 2 * batch");
    }

    const int n = input.size() / (2 * batch);  // Длина каждого сигнала в комплексных числах
    const int total_complex = n * batch;
    const size_t bytes = sizeof(hipfftComplex) * total_complex;

    // Выделение памяти на устройстве
    hipfftComplex* d_data = nullptr;
    hipError_t cudaStatus = hipMalloc(&d_data, bytes);
    if (cudaStatus != hipSuccess) {
        throw std::runtime_error("CUDA memory allocation failed");
    }

    // Копирование данных на устройство
    cudaStatus = hipMemcpy(d_data, input.data(), bytes, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        hipFree(d_data);
        throw std::runtime_error("CUDA memcpy to device failed");
    }

    // Создание плана cuFFT
    hipfftHandle plan;
    hipfftResult cufftStatus = hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);
    if (cufftStatus != HIPFFT_SUCCESS) {
        hipFree(d_data);
        throw std::runtime_error("cuFFT plan creation failed");
    }

    try {
        // Прямое БПФ
        cufftStatus = hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
        if (cufftStatus != HIPFFT_SUCCESS) {
            throw std::runtime_error("Forward FFT failed");
        }

        // Обратное БПФ
        cufftStatus = hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);
        if (cufftStatus != HIPFFT_SUCCESS) {
            throw std::runtime_error("Inverse FFT failed");
        }

        // Нормализация
        const float inv_n = 1.0f / static_cast<float>(n);
        const int blockSize = 256;
        const int gridSize = (total_complex + blockSize - 1) / blockSize;
        
        normalize_kernel<<<gridSize, blockSize>>>(d_data, total_complex, inv_n);
        
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("Kernel launch failed");
        }

        // Синхронизация
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("CUDA synchronization failed");
        }

        // Копирование результата обратно
        std::vector<float> output(input.size());
        cudaStatus = hipMemcpy(output.data(), d_data, bytes, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("CUDA memcpy to host failed");
        }

        // Освобождение ресурсов
        hipfftDestroy(plan);
        hipFree(d_data);

        return output;

    } catch (const std::exception& e) {
        // Освобождение ресурсов при ошибке
        hipfftDestroy(plan);
        hipFree(d_data);
        throw;
    }
}