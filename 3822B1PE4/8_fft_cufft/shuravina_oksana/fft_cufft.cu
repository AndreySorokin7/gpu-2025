#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>
#include <vector>

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    // Проверка входных данных
    if (input.size() % (2 * batch) != 0) {
        throw std::invalid_argument("Input size must be divisible by 2 * batch");
    }
    
    int n = input.size() / (2 * batch);  // Длина каждого сигнала в комплексных числах
    
    if (n == 0) {
        throw std::invalid_argument("Signal length cannot be zero");
    }

    hipfftResult cufftStatus;
    hipError_t cudaStatus;
    
    // Создание плана для прямого БПФ
    hipfftHandle planForward;
    cufftStatus = hipfftPlan1d(&planForward, n, HIPFFT_C2C, batch);
    if (cufftStatus != HIPFFT_SUCCESS) {
        throw std::runtime_error("Failed to create forward FFT plan");
    }
    
    // Создание плана для обратного БПФ
    hipfftHandle planInverse;
    cufftStatus = hipfftPlan1d(&planInverse, n, HIPFFT_C2C, batch);
    if (cufftStatus != HIPFFT_SUCCESS) {
        hipfftDestroy(planForward);
        throw std::runtime_error("Failed to create inverse FFT plan");
    }

    // Выделение памяти на устройстве
    hipfftComplex *d_input, *d_fft, *d_result;
    size_t complexSize = n * batch * sizeof(hipfftComplex);
    
    cudaStatus = hipMalloc(&d_input, complexSize);
    if (cudaStatus != hipSuccess) {
        hipfftDestroy(planForward);
        hipfftDestroy(planInverse);
        throw std::runtime_error("CUDA memory allocation for input failed");
    }
    
    cudaStatus = hipMalloc(&d_fft, complexSize);
    if (cudaStatus != hipSuccess) {
        hipFree(d_input);
        hipfftDestroy(planForward);
        hipfftDestroy(planInverse);
        throw std::runtime_error("CUDA memory allocation for FFT failed");
    }
    
    cudaStatus = hipMalloc(&d_result, complexSize);
    if (cudaStatus != hipSuccess) {
        hipFree(d_input);
        hipFree(d_fft);
        hipfftDestroy(planForward);
        hipfftDestroy(planInverse);
        throw std::runtime_error("CUDA memory allocation for result failed");
    }

    try {
        // Копирование входных данных на устройство
        // Преобразование из пар float в hipfftComplex
        cudaStatus = hipMemcpy(d_input, input.data(), complexSize, hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("Failed to copy input data to device");
        }
        
        // Прямое преобразование Фурье
        cufftStatus = hipfftExecC2C(planForward, d_input, d_fft, HIPFFT_FORWARD);
        if (cufftStatus != HIPFFT_SUCCESS) {
            throw std::runtime_error("Forward FFT execution failed");
        }
        
        // Обратное преобразование Фурье
        cufftStatus = hipfftExecC2C(planInverse, d_fft, d_result, HIPFFT_BACKWARD);
        if (cufftStatus != HIPFFT_SUCCESS) {
            throw std::runtime_error("Inverse FFT execution failed");
        }
        
        // Нормализация результата на устройстве
        float scale = 1.0f / n;
        
        // Запускаем ядро для нормализации
        int totalElements = n * batch;
        int blockSize = 256;
        int numBlocks = (totalElements + blockSize - 1) / blockSize;
        
        // Kernel для нормализации
        auto normalizeKernel = [](hipfftComplex* data, float scale, int n) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < n) {
                data[idx].x *= scale;
                data[idx].y *= scale;
            }
        };
        
        normalizeKernel<<<numBlocks, blockSize>>>(d_result, scale, totalElements);
        
        // Проверка ошибок ядра
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("Normalization kernel failed");
        }
        
        // Синхронизация для обеспечения завершения всех операций
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("CUDA device synchronization failed");
        }
        
        // Копирование результата обратно на хост
        std::vector<float> output(input.size());
        cudaStatus = hipMemcpy(output.data(), d_result, complexSize, hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            throw std::runtime_error("Failed to copy result data from device");
        }
        
        // Освобождение ресурсов
        hipFree(d_input);
        hipFree(d_fft);
        hipFree(d_result);
        hipfftDestroy(planForward);
        hipfftDestroy(planInverse);
        
        return output;

    } catch (const std::exception& e) {
        // Освобождение ресурсов в случае ошибки
        hipFree(d_input);
        hipFree(d_fft);
        hipFree(d_result);
        hipfftDestroy(planForward);
        hipfftDestroy(planInverse);
        throw;
    }
}