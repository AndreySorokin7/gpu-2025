﻿#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "gemm_cublas.h"

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b, int n) {
  const size_t elems = static_cast<size_t>(n) * n;
  const size_t bytes = elems * sizeof(float);

  float *dA = nullptr, *dB = nullptr, *dC = nullptr;
  hipMalloc(&dA, bytes);
  hipMalloc(&dB, bytes);
  hipMalloc(&dC, bytes);

  hipMemcpy(dA, a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(dB, b.data(), bytes, hipMemcpyHostToDevice);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  const float alpha = 1.0f;
  const float beta = 0.0f;

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
              n,             
              n,           
              n,             
              &alpha, dB, n, 
              dA, n,    
              &beta, dC, n
  );


  hipDeviceSynchronize();


  std::vector<float> c(elems);
  hipMemcpy(c.data(), dC, bytes, hipMemcpyDeviceToHost);


  hipblasDestroy(handle);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);

  return c;
}
